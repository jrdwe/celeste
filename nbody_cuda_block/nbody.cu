#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <getopt.h>
#include <string.h>

#include "nbody.cuh"
#include "parsing.cuh"

#define BLOCK_SIZE 128
#define NUM_STRUCT_FIELDS 7
#define STRUCT_FIELD_MASS_INDEX 6
#define MIN_ARGUMENTS 5
#define DAMPEN 1e-9
#define ACCEPTABLE_VARIANCE 1e-1
#define DEFAULT_SIZE 50
#define DEFAULT_LOWER_BOUND -50
#define DEFAULT_UPPER_BOUND 50

void randomiseCelestialBodies(double* body, int size, double lower, double upper) {
  // generate a new seed
  time_t timenow;
  srand((unsigned) time(&timenow));

  // generate double doubles within the provided range
  for (int i = 0; i < size; ++i) {
    double s = (double) rand() / RAND_MAX;

    if (i % NUM_STRUCT_FIELDS == STRUCT_FIELD_MASS_INDEX) {
      // ensures mass isn't negative
      body[i] = upper * s;
    } else {
      body[i] = (upper - lower) * s + lower;
    }
  }
}

__global__
void step(struct body* bo, int num, float dt) {
  // calculate thread index
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < num) {
    double ax = 0.0, ay = 0.0, az = 0.0;

    // define shared memory arrays for x, y, z, & mass values
    __shared__ double cx[BLOCK_SIZE], 
                      cy[BLOCK_SIZE], 
                      cz[BLOCK_SIZE], 
                      cm[BLOCK_SIZE];

    for (int set = 0; set < num; set += BLOCK_SIZE) {
      // each thread loads value into the shared memory for the block
      cx[set + threadIdx.x] = bo[i].x; 
      cy[set + threadIdx.x] = bo[i].y; 
      cz[set + threadIdx.x] = bo[i].z;
      cm[set + threadIdx.x] = bo[i].mass; 
      
      // sync all threads before shared memory access 
      __syncthreads();

      for (int j = 0; j < BLOCK_SIZE; ++j) {
  
        // calculate relative distances
        double dx   = cx[j] - bo[i].x;
        double dy   = cy[j] - bo[i].y;
        double dz   = cz[j] - bo[i].z;

        // dampen ensures we don't take the sqrt of zero
        double dist = sqrt((double) ((dx * dx) + (dy * dy) + (dz * dz) + DAMPEN));

        // calculate gravitional magnitude
        double magn = GCONST * ((bo[i].mass * cm[j])/(dist * dist));
  
        // magnitude * direction
        ax         += (((dx / dist) * magn) / bo[i].mass);
        ay         += (((dy / dist) * magn) / bo[i].mass);
        az         += (((dz / dist) * magn) / bo[i].mass);
      }
  
      // sync all threads before next cache load 
      __syncthreads();
    }
    
    // update velocity values
    bo[i].velocity_x += dt * ax;
    bo[i].velocity_y += dt * ay;
    bo[i].velocity_z += dt * az;
  }
}

double energy(struct body* bo, int num) {

  double e = 0.0f;
  for (int i = 0; i < num; ++i) {
    double matsqd = (double) (bo[i].velocity_x * bo[i].velocity_x)
        + (bo[i].velocity_y * bo[i].velocity_y) 
        + (bo[i].velocity_z * bo[i].velocity_z);
    double first  = (bo[i].mass * matsqd) / 2;
    double second = 0.0f;

    for (int j = i + 1; j < num; ++j) {
      double dx   = bo[j].x - bo[i].x;
      double dy   = bo[j].y - bo[i].y;
      double dz   = bo[j].z - bo[i].z;
      double dist = sqrtf(((dx * dx) + (dy*dy) + (dz*dz)));
      second     += (GCONST * bo[i].mass * bo[j].mass) / dist;
    }

    e += first - second;
  }

  return e;
}

int main(int argc, char** argv) {

  // check required parameters are provided
  if (argc < MIN_ARGUMENTS) {
    printf("usage: ./nbody <iters> <dt> ((-b <bodies> -l <lowerbound> -u <upperbound>) | -f <filename>) <testmode> \n");
    exit(EXIT_FAILURE);
  }

  // variable declaration
  struct body* bodies;
  struct body* d_bodies;
  double ebefore;
  double eafter;
  clock_t tic;
  clock_t toc;
  char* fstring       = NULL;
  int f               = 0;
  int testmode        = 0;
  int size            = DEFAULT_SIZE;
  double upper        = DEFAULT_UPPER_BOUND;
  double lower        = DEFAULT_LOWER_BOUND;
  const int iteration = atoi(argv[1]);
  const float dt      = atof(argv[2]);

  // parse all flag information
  parseParameters(argv, argc, &f, &size, &upper, &lower, &testmode, &fstring);

  // get size by reading file
  if (f) 
    size = countLines(fstring);

  // malloc required size
  bodies = (struct body*) malloc(sizeof(struct body) * size);
  if (!bodies) { 
    fprintf(stderr, "Unable to malloc requested bodies"); 
    exit(EXIT_FAILURE); 
  }

  if (f) {
    // read in data from provided file
    readDataFromFile(fstring, (double*) bodies);
  } else {
    // randomise all bodies
    randomiseCelestialBodies((double*) bodies, NUM_STRUCT_FIELDS * size, lower, upper);
  }

  // allocate memory for GPU 
  hipMalloc(&d_bodies, sizeof(struct body) * size);

  // calculate the required number of blocks
  int nblocks = (size + BLOCK_SIZE) / BLOCK_SIZE;

  if (testmode) {
    // calculate the energy within the system prior
    ebefore = energy(bodies, size);
    
    // record the starting time
    toc = clock();
  }

  for (int k = 0; k < iteration; ++k) {

    // copy memory over to the GPU
    hipMemcpy(d_bodies, bodies, sizeof(struct body) * size, hipMemcpyHostToDevice);

    // execute GPU kernel function
    step<<<nblocks, BLOCK_SIZE>>>(d_bodies, size, dt);
    
    // block CPU execution until all previously issued commands on the device have completed
    hipDeviceSynchronize();

    // copy memory back from the GPU
    hipMemcpy(bodies, d_bodies, sizeof(struct body) * size, hipMemcpyDeviceToHost);

    // update the x & y positions for each data point
    for (int i = 0; i < size; ++i) {
      bodies[i].x += bodies[i].velocity_x * dt;
      bodies[i].y += bodies[i].velocity_y * dt;
      bodies[i].z += bodies[i].velocity_z * dt;
    }
  }

  // free memory for GPU
  hipFree(d_bodies);

  if (testmode) {
    // calculate the energy within the system after
    eafter = energy(bodies, size);

    // calculate the finish time
    tic = clock();
    double time_taken = (double)(tic - toc) / CLOCKS_PER_SEC;
   
    // output time taken to execute function
    printf("Time taken for %d iterations with %d bodies was %f\n", iteration, size, time_taken);

    // check if the energy within the system is acceptable
    if (fabs(ebefore - eafter) > ACCEPTABLE_VARIANCE) {
      printf("Energy did not match %f vs %f\n", ebefore, eafter);
      printf("Absolute Error was %f & Relative Error was %f%%\n", (eafter - ebefore), (fabs(eafter - ebefore) / ebefore) * 100);
    }
  }

  // free allocated memory
  free(bodies);
  return EXIT_SUCCESS;
}
